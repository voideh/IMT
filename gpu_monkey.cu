#include "hip/hip_runtime.h"
#include <stdio.h>
#include "gpu_monkey.h"
#include "/usr/local/cuda/include/hiprand.h"
#include "/usr/local/cuda/include/hiprand/hiprand_kernel.h"

/**
    The monkey simulator that takes a string and repeatedly creates random
    strings of equal length, until it matches the input string.
*/

void generateMonkey(char* genInput, int wordc)
{
	
//	printf("words: %s\n", genInput);
	/* Device pointers */
	int* d_key;
	int* d_key2;
	int* d_toklen;

	/* Host pointers*/
	int toklen[wordc];
	int key[wordc];
	int key2[wordc];

	int x = 0;

	/* Initialiizing array to 0 for both arrays of keys */
	while(x < wordc)
	{
		key[x] = 0;
		key2[x] = 0;
		x++;
	}

	char* saveWord = genInput;
	int tokiter = 0;
	/* strtok_r is used to parse the input for each word */
	char* token = strtok_r(genInput, "\t\n ", &saveWord);
	while(token != NULL)
	{
		/* Storing length of the word*/
		toklen[tokiter] = static_cast< int > (strlen(token));

		int i = 0;
		
		while(i < toklen[tokiter])
		{
			/* Generate a key for each word, based upon character and
			   order that characters appears...
			   key = (character position + 1) * character + character */


			// Calculates the key value for every lower case letter,
			//	when given a lowercase letter
			if(token[i] >= 97  &&  token[i] < 123)
			{
				key[tokiter] += (i + 1) * token[i] + token[i];
				key2[tokiter] += token[i];
				i++;
			}
			// Calculates the key value for every lower case letter,
			//	when given an uppercase letter
			else if(token[i] >= 65  &&  token[i] < 91)
			{
				key[tokiter] += (i + 1) * (token[i] + 32) +  (token[i] + 32);
				key2[tokiter] += (token[i] + 32);
				i++;
			}

			// Handles the appearence of special characters
			else
			{
				//loops to eliminate the special character from the word, for matching purposes
				int j;
				for(j = i; j < toklen[tokiter]; j++)
				{
					token[i] = token[i+1];
				}
				toklen[tokiter] = toklen[tokiter] - 1;
			}
		}
		tokiter++;                             
		token = strtok_r(NULL, "\n ", &saveWord);  
	}                                          

	printf("Word Count: %d\n", wordc);

	/* Allocate and memcopy first key */
	if(hipMalloc((void**)&d_key, sizeof(int) * wordc) != hipSuccess)
	{
		printf("Couldn't allocate memory for d_key\n");
		return;
	}

	if(hipMemcpy(d_key, key, sizeof(int) * wordc,  hipMemcpyHostToDevice) != hipSuccess)
	{
		printf("Couldn't allocate memory for d_key\n");
		return;
	}

	/* Allocate and memcopy second key */
	if(hipMalloc((void**)&d_key2, sizeof(int) * wordc) != hipSuccess)
	{
		printf("Couldn't allocate memory for d_key\n");
		return;
	}

	if(hipMemcpy(d_key2, key2, sizeof(int) * wordc,  hipMemcpyHostToDevice) != hipSuccess)
	{
		printf("Couldn't allocate memory for d_key\n");
		return;
	}

	/* Allocate and memcopy length of word */
	if(hipMalloc((void**)&d_toklen, sizeof(int) * wordc) != hipSuccess)
	{
		printf("Couldn't allocate memory for d_key\n");
		return;
	}

	if(hipMemcpy(d_toklen, toklen, sizeof(int) * wordc,  hipMemcpyHostToDevice) != hipSuccess)
	{
		printf("Couldn't allocate memory for d_toklen\n");
		return;
	}

	/* Runtime Params:	"wordc" for the amount of blocks needed 
	   					"32" for the ammount of threads for each block
	   Arguments: 		"toklen" for the length of a word
	   					"d_key" for the generated key to check against
						"d_key2" for the generated key to check against
						"seed" to seed our cudarand function
						"wordc" to get the word count
	*/	

	/* rand used to seed hiprand in the kernel */
	srand(time(NULL));
	unsigned int seed = rand();
	monkey<<<wordc, 32>>>(d_toklen, d_key, d_key2, seed, wordc);

	/* Checking for last error*/
	hipError_t err = hipGetLastError();
	if (err != hipSuccess) 
		    printf("Error: %s\n", hipGetErrorString(err));

	/* Freeing allocated memory from device */
	hipFree(d_key);
	hipFree(d_toklen);
	hipFree(d_key2);
}

__global__ void monkey(int* toklen, int* d_key, int* d_key2, unsigned int seed, int wordc)
{

	int id = blockIdx.x * blockDim.x + threadIdx.x;

	/* Condition was to check if thread id was between the largest thread id of the last block 
	   and the length of the word */
	if(id < blockIdx.x * blockDim.x + toklen[blockIdx.x] && id >= blockIdx.x * blockDim.x)
	{
		/*

		    	match       = simulates a boolean value. becomes 1 when the monkey
							matches the original input.
				count       = counts the number of attempts the monkey made
				key			= the random string created by the monkey
				random      = the random number generated
							0-25 correspond to the 26 letters of the alphabet
							26 corresponds to the space on the keyboard	
																				*/
		hiprandState_t state;

		/* Seed for random number for each thread to
		   choose a unique key from the assigned character*/
		hiprand_init(seed*(id + 1),0,0,&state);
		
		/* Keygen is for the first array of keys
		   Keygen2 is for the second array of keys
		   cumalitivekey 1 and 2 are the summation of
		   keygen 1 and 2 respectively */
		__shared__ int keygen[32];
		__shared__ int keygen2[32];
		__shared__ int cumulativekey;
		__shared__ int cumulativekey2;
		__shared__ int match;

		/* Initializing both arrays of keys to 0 */
		if(id == (blockIdx.x * blockDim.x))
		{
			match = 0;
			for(int x = 0; x < 32; x++)
			{
				keygen[x] = 0;
				keygen2[x] = 0;
			}
		}

		int count = 0;
		int random;
			/*
		    	Following loop creates a random string of appropriate length.
				The way that the cascading IF statements are set up optimize the
				random choosing of a letter.
					-   It takes 5 "decisions" to choose a letter based on the random number
					-   This process if done linearly, could take up to 26 "decisions" */
		int i = 0;
		while(i++ < 1000000000)   // Loops until the all threads matches the original input
		{
			if(id == (blockIdx.x * blockDim.x))
			{
				cumulativekey2 = 0;
				cumulativekey =0;
				count++;
			}	

			__syncthreads();
				random = hiprand(&state) % 10000;

				if(random < 8998)
				{
					if(random < 6128)
					{
						if(random < 3478)
						{
							if(random < 1965)
							{
								if(random < 1116)
								{
									keygen[threadIdx.x] = (threadIdx.x + 1) * 101 + 101;   //e
									keygen2[threadIdx.x] = 101;   //e
								}
								else
								{
									keygen[threadIdx.x] = (threadIdx.x + 1) * 97 + 97;    //a
									keygen2[threadIdx.x] = 97;   //a
								}
							}
							else
							{
								if(random < 2723)
								{
									keygen[threadIdx.x] = (threadIdx.x + 1) * 114 + 114;   //r
									keygen2[threadIdx.x] = 114;   //r
								}
								else
								{
									keygen[threadIdx.x] = (threadIdx.x + 1) * 105 + 105;   //i
									keygen2[threadIdx.x] = 105;   //i
								}
							}
						}
						else
						{
							if(random < 4889)
							{
								if(random < 4194)
								{
									keygen[threadIdx.x] = (threadIdx.x + 1) * 111 + 111;   //o
									keygen2[threadIdx.x] = 111;   //o
								}
								else
								{
									keygen[threadIdx.x] = (threadIdx.x + 1) * 116 + 116;   //t
									keygen2[threadIdx.x] = 116;   //t
								}
							}
							else
							{
								if(random < 5555)
								{
									keygen[threadIdx.x] = (threadIdx.x + 1) * 110 + 110;   //n
									keygen2[threadIdx.x] = 110;   //n
								}
								else
								{
									keygen[threadIdx.x] = (threadIdx.x + 1) * 115 + 115;   //s
									keygen2[threadIdx.x] = 115;   //s
								}
							}
						}
					}
					else
					{
						if(random < 7832)
						{
							if(random < 7131)
							{
								if(random < 6677)
								{
									keygen[threadIdx.x] = (threadIdx.x + 1) * 108 + 108;   //l
									keygen2[threadIdx.x] = 108;   //l
								}
								else
								{
									keygen[threadIdx.x] = (threadIdx.x + 1) * 99 + 99;    //c
									keygen2[threadIdx.x] = 99;   //c
								}
							}
							else
							{
								if(random < 7494)
								{
									keygen[threadIdx.x] = (threadIdx.x + 1) * 117 + 117;   //u
									keygen2[threadIdx.x] = 117;   //u
								}
								else
								{
									keygen[threadIdx.x] = (threadIdx.x + 1) * 100 + 100;   //d
									keygen2[threadIdx.x] = 100;   //d
								}
							}
						}
						else
						{
							if(random < 8450)
							{
								if(random < 8149)
								{
									keygen[threadIdx.x] = (threadIdx.x + 1) * 112 + 112;   //p
									keygen2[threadIdx.x] = 112;   //p
								}
								else
								{
									keygen[threadIdx.x] = (threadIdx.x + 1) * 109 + 109;   //m
									keygen2[threadIdx.x] = 109;   //m
								}
							}
							else
							{
								if(random < 8751)
								{
									keygen[threadIdx.x] = (threadIdx.x + 1) * 104 + 104;   //h
									keygen2[threadIdx.x] = 104;   //h
								}
								else
								{
									keygen[threadIdx.x] = (threadIdx.x + 1) * 103 + 103;   //g
									keygen2[threadIdx.x] = 103;   //g
								}
							}
						}
					}
				}
				else
				{
					if(random < 9960)
					{
						if(random < 9693)
						{
							if(random < 9386)
							{
								if(random < 9206)
								{
									keygen[threadIdx.x] = (threadIdx.x + 1) * 98 + 98;    //b
									keygen2[threadIdx.x] = 98;   //b
								}
								else
								{
									keygen[threadIdx.x] = (threadIdx.x + 1) * 102 + 102;   //f
									keygen2[threadIdx.x] = 102;   //f
								}
							}
							else
							{
								if(random < 9564)
								{
									keygen[threadIdx.x] = (threadIdx.x + 1) * 121 + 121;   //y
									keygen2[threadIdx.x] = 121;   //y
								}
								else
								{
									keygen[threadIdx.x] = (threadIdx.x + 1) * 119 + 119;   //w
									keygen2[threadIdx.x] = 119;   //w
								}
							}
						}
						else
						{
							if(random < 9904)
							{
								if(random < 9803)
								{
									keygen[threadIdx.x] = (threadIdx.x + 1) * 107 + 107;   //k
									keygen2[threadIdx.x] = 107;   //k
								}
								else
								{
									keygen[threadIdx.x] = (threadIdx.x + 1) * 118 + 118;   //v
									keygen2[threadIdx.x] = 118;   //v
								}
							}
							else
							{
								if(random < 9933)
								{
									keygen[threadIdx.x] = (threadIdx.x + 1) * 120 + 120;   //x
									keygen2[threadIdx.x] = 120;   //x
								}
								else
								{
									keygen[threadIdx.x] = (threadIdx.x + 1) * 122 + 122;   //z
									keygen2[threadIdx.x] = 122;   //z
								}
							}
						}
					}
					else
					{
						if(random < 9980)
						{
							keygen[threadIdx.x] = (threadIdx.x + 1) * 106 + 106;   //j
							keygen2[threadIdx.x] = 106;   //j
						}
						else
						{
							keygen[threadIdx.x] = (threadIdx.x + 1) * 113 + 113 ;   //q
							keygen2[threadIdx.x] = 113;   //q
						}
					}
				}
				__syncthreads();

				/* Summation of the keys accumulated above */
				if(id == (blockIdx.x * blockDim.x))
				{
					for(int s = 0; s < toklen[blockIdx.x]; s++)
					{
						cumulativekey = cumulativekey + keygen[s];
						cumulativekey2 += keygen2[s];
					}
				}

				// Compare the keygen with the original input
				if(cumulativekey == d_key[blockIdx.x] && cumulativekey2 == d_key2[blockIdx.x]
						&& id == (blockIdx.x * blockDim.x))
					match = 1;
				__syncthreads();

				/* match is set if word was found 
				   and block leaves loop*/
				if(match)
					break;
				__syncthreads();
			}
	__syncthreads();
	}
}

